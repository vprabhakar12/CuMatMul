#include <hip/hip_runtime.h>
#include <iostream>
// using namespace std;

#define BLOCK_SIZE 32

__global__ void matmul_naive(const float* A, const float* B, float* C,
                             int M, int K, int N) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // x = row index (first dimension of output C)
    const int y = blockIdx.y * blockDim.y + threadIdx.y;  // y = col index (second dimension of output C)

    if (x < M && y < N) {
        float sum = 0.0f;
        for (int i = 0; i < K; ++i) {
            sum += A[x * K + i] * B[i * N + y];
        }
        C[x * N + y] = sum;
    }
    // C[0] = 5;
}

extern "C" void launch_naive(float* A, float* B, float* C, int M, int K, int N) {
    float *d_A, *d_B, *d_C;
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((M + BLOCK_SIZE-1) / BLOCK_SIZE, (N + BLOCK_SIZE-1) / BLOCK_SIZE);

    matmul_naive<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, K, N);

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    // std::cerr <<" value " << *C << std::endl;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

__global__ void matmul_gmemcoal(const float* A, const float* B, float* C,
                             int M, int K, int N) {
    const int x = blockIdx.x * BLOCK_SIZE + threadIdx.x/BLOCK_SIZE;  // x = col index (2nd dimension of output C)
    const int y = blockIdx.y * BLOCK_SIZE + threadIdx.x%BLOCK_SIZE;  // y = row index (1st dimension of output C)

    if (x < M && y < N) {
        float sum = 0.0f;
        for (int i = 0; i < K; ++i) {
            sum += A[x * K + i] * B[i * N + y];
        }
        C[x * N + y] = sum;
    }
    // C[0] = 5;
}

extern "C" void launch_gmemcoal(float* A, float* B, float* C, int M, int K, int N) {
    float *d_A, *d_B, *d_C;
    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size_B, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE * BLOCK_SIZE);
    dim3 gridSize((M + BLOCK_SIZE-1) / BLOCK_SIZE, (N + BLOCK_SIZE-1) / BLOCK_SIZE);

    matmul_gmemcoal<<<gridSize, blockSize>>>(d_A, d_B, d_C, M, K, N);

    hipMemcpy(C, d_C, size_C, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

#define TILE_SIZE 16

__global__ void matmul_tiled(const float* A, const float* B, float* C, int N) {
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0.0f;

    for (int tile = 0; tile < (N + TILE_SIZE - 1) / TILE_SIZE; ++tile) {
        if (row < N && tile * TILE_SIZE + threadIdx.x < N)
            tileA[threadIdx.y][threadIdx.x] = A[row * N + tile * TILE_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && tile * TILE_SIZE + threadIdx.y < N)
            tileB[threadIdx.y][threadIdx.x] = B[(tile * TILE_SIZE + threadIdx.y) * N + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

extern "C" void launch_tiled(float* A, float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 blockSize(TILE_SIZE, TILE_SIZE);
    dim3 gridSize((N + TILE_SIZE - 1) / TILE_SIZE, (N + TILE_SIZE - 1) / TILE_SIZE);

    matmul_tiled<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}